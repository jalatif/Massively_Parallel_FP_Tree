
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

#define n_items_in_transaction 128
#define max_items_in_transaction 100000

int main(int argc, char**argv) {

    //cudaError_t cuda_ret;

    FILE *fp = fopen("/home/manshu/UIUC/CS 412 - Data Mining/data-assign3/data-assign3/topic-0.txt", "r");
    char *line = NULL;
    size_t len = 0;
    ssize_t read;
    if (fp == NULL){
    	printf("Can't read file");
    	exit(0);
    }
    int lines = 0;
    int count = 0;
    char *ln;
    unsigned short int *trans = NULL;
    unsigned short int **transactions = NULL;
    transactions = (unsigned short int **) malloc(max_items_in_transaction * sizeof(unsigned short int *));

    while ((read = getline(&line, &len, fp)) != -1){
    	// printf("%s", line);
		trans = (unsigned short int *) malloc(n_items_in_transaction * sizeof(unsigned short int));
    	
    	count = 0;

    	ln = strtok(line, " ");
    	if (ln != NULL){
    			//unsigned short int a = (unsigned short int) strtoul(ln, NULL, 0);
    			trans[count++] = (unsigned short int) strtoul(ln, NULL, 0);
    	}
    	
    	while (ln != NULL){
    		// printf("%s ", ln);
    		ln = strtok(NULL, " ");
    		if (ln != NULL){
    			trans[count++] = (unsigned short int) strtoul(ln, NULL, 0);
    		}

    	}
    	trans[count] = NULL;

    	if (trans == NULL)
    	transactions[lines] = trans;
    	int j = 0;
    	unsigned short int *temp = transactions[lines];
   		while(temp != NULL && temp[j] != NULL){
   			printf("%hu, ", temp[j]);
   			j++;
   		}
   		printf("\n");
   		lines++;
    }
    fclose(fp);
    transactions[lines] = NULL;

    for(int i = 0; i < lines; i++){
    	int j = 0;
    	unsigned short int *temp = transactions[i];
   		while(temp != NULL && temp[j] != NULL){
   			printf("%x, ", temp[j]);
   			j++;
   		}
   		//printf("\n");
    }

    printf("%d", lines);
    int j = 0;
    unsigned short int *temp = transactions[0];
   		while(temp != NULL && temp[j] != NULL){
   			printf("%x, ", temp[j]);
   			j++;
   	}
    // float* A_h = (float*) malloc( sizeof(float)*n );
    // for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    // float* B_h = (float*) malloc( sizeof(float)*n );
    // for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    // float* C_h = (float*) malloc( sizeof(float)*n );

    // printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    // printf("Allocating device variables...\n"); fflush(stdout);

    // float *A_d, *B_d, *C_d;
    // int size = n * sizeof(float);

    // cudaError_t status;
    // status = cudaMalloc((void **) &A_d, size);
    // if (status != cudaSuccess){
    // 	printf("%s in %s at line %d\n", cudaGetErrorString(status), __FILE__, __LINE__);
    // 	exit(EXIT_FAILURE);
    // }
    // status = cudaMalloc((void **) &B_d, size);
    // if (status != cudaSuccess){
    // 	printf("%s in %s at line %d\n", cudaGetErrorString(status), __FILE__, __LINE__);
    // 	exit(EXIT_FAILURE);
    // }
    // status = cudaMalloc((void **) &C_d, size);
    // if (status != cudaSuccess){
    // 	printf("%s in %s at line %d\n", cudaGetErrorString(status), __FILE__, __LINE__);
    // 	exit(EXIT_FAILURE);
    // }

    // cudaDeviceSynchronize();

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    

    //INSERT CODE HERE

 //    cudaMemcpy(A_d, A_h, size, cudaMemcpyHostToDevice);
	// cudaMemcpy(B_d, B_h, size, cudaMemcpyHostToDevice);

 //    cudaDeviceSynchronize();
    

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);

    //INSERT CODE HERE

    // dim3 DimGrid((n-1)/256 + 1, 1, 1);
    // dim3 DimBlock(256, 1, 1);

    //vecAddKernel<<<DimGrid, DimBlock>>>(A_d, B_d, C_d, n);



 //    cuda_ret = cudaDeviceSynchronize();
	// if(cuda_ret != cudaSuccess) {
 //        printf("Unable to launch kernel");
 //        exit(EXIT_FAILURE);
 //    }

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    

    //INSERT CODE HERE

    // cudaMemcpy(C_h, C_d, size, cudaMemcpyDeviceToHost);

    // cudaDeviceSynchronize();

    // Verify correctness -----------------------------------------------------

    // printf("Verifying results..."); fflush(stdout);

    // Free memory ------------------------------------------------------------

    // free(A_h);
    // free(B_h);
    // free(C_h);

    //INSERT CODE HERE

    // cudaFree(A_d);
    // cudaFree(B_d);
    // cudaFree(C_d);


    return 0;

}
