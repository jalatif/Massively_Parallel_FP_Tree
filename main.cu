#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

	//File read into datastructures

    FILE *fp = fopen("topic-0.txt", "r");
    if (fp == NULL){
    	printf("Can't read file");
    	exit(0);
    }
    char *line = NULL;
    size_t len = 0;
    ssize_t read;
    unsigned int lines = 0;
    unsigned int count = 0;
    char *ln, *nptr;
    
    unsigned int *transactions = NULL;
    unsigned int *trans_offset = NULL;
    unsigned int *flist = NULL;

    unsigned int element_id = 0;
    unsigned int check_null = 0;
    
    transactions = (unsigned int *) malloc(max_num_of_transaction * max_items_in_transaction * sizeof(unsigned int));
    trans_offset = (unsigned int *) malloc((max_num_of_transaction + 1) * sizeof(unsigned int));
    flist = (unsigned int *) malloc(max_unique_items * sizeof(unsigned int));

    trans_offset[0] = 0;

    while ((read = getline(&line, &len, fp)) != -1){
    	
    	count = 0;

    	ln = strtok(line, " ");
    	if (ln != NULL){
    			//unsigned int a = (unsigned int) strtoul(ln, NULL, 0);
    			transactions[element_id++] = (unsigned int) strtoul(ln, NULL, 0);
    			count++;
    	}
    	
    	while (ln != NULL){
    		// printf("%s ", ln);
    		ln = strtok(NULL, " ");
    		if (ln != NULL){
    			check_null = (unsigned int) strtoul(ln, &nptr, 0);
    			if (strcmp(nptr, ln) != 0){
    				transactions[element_id++] = check_null;
    				count++;
    			}
    		}

    	}

    	trans_offset[lines + 1] = trans_offset[lines] + count;

   		lines++;
    }
    fclose(fp);

    trans_offset[lines] = NULL;
    //transactions[element_id] = NULL;

    unsigned int num_items_in_transactions = element_id;
    unsigned int num_transactions = lines;

    // for (int i = 0; i < num_transactions; i++){
    // 	int item_ends = 0;
    // 	if (i == (num_transactions - 1)){
    // 		item_ends = num_items_in_transactions;
    // 	}else{
    // 		item_ends = trans_offset[i+1];
    // 	}
    // 	for (int j = trans_offset[i]; j < item_ends; j++)
    // 		printf("%hu ", transactions[j]);
    // 	printf("\n");
    // }

    printf("Number of Transactions = %d\n", lines);
  
    /////////////////////////////////////////////////////////////////////////////////////
    /////////////////////// Device Variables Initializations ///////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////

	Timer timer;
	hipError_t cuda_ret;
	dim3 grid_dim, block_dim;

    unsigned int *d_transactions;
    unsigned int *d_trans_offsets;
    unsigned int *d_flist;
    
    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMalloc((void**)&d_transactions, num_items_in_transactions * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    cuda_ret = hipMalloc((void**)&d_trans_offsets, num_transactions * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
   	cuda_ret = hipMalloc((void**)&d_flist, max_unique_items * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
	
	cuda_ret = hipMemset(d_flist, 0, max_unique_items * sizeof(unsigned int));
    if(cuda_ret != hipSuccess) FATAL("Unable to set device memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------
	
	printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(d_transactions, transactions, num_items_in_transactions * sizeof(unsigned int),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

	cuda_ret = hipMemcpy(d_trans_offsets, trans_offset, num_transactions * sizeof(unsigned int),
        hipMemcpyHostToDevice);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device");

    
    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    block_dim.x = BLOCK_SIZE; 
    block_dim.y = 1; block_dim.z = 1;
    
    grid_dim.x = ceil(num_transactions / (1.0 * BLOCK_SIZE)); 
    grid_dim.y = 1; grid_dim.z = 1;  

	//size_t private_flist_size = max_unique_items * sizeof(unsigned int);
    makeFlist<<<grid_dim, block_dim>>>(d_trans_offsets, d_transactions, d_flist, num_transactions, num_items_in_transactions);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

 	// Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    cuda_ret = hipMemcpy(flist, d_flist, max_unique_items * sizeof(unsigned int),
        hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    //#if TEST_MODE
    printf("\nResult:\n");
    for(unsigned int i = 0; i < max_unique_items; ++i) {
        printf("Item %u: %u frequency\n", i, flist[i]);
    }
    //#endif

    // Free memory ------------------------------------------------------------

    free(trans_offset);
    free(transactions);

    hipFree(d_trans_offsets);
    hipFree(d_transactions);
    
    return 0;

}
