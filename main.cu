
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

#define max_items_in_transaction 128
#define max_num_of_transaction 100000

int main(int argc, char**argv) {

	//File read into datastructures

    FILE *fp = fopen("/home/manshu/UIUC/CS 412 - Data Mining/data-assign3/data-assign3/topic-0.txt", "r");
    char *line = NULL;
    size_t len = 0;
    ssize_t read;
    if (fp == NULL){
    	printf("Can't read file");
    	exit(0);
    }
    int lines = 0;
    int count = 0;
    char *ln;
    unsigned short int **transactions = NULL;
    unsigned short int **
    transactions = (unsigned short int **) malloc(max_num_of_transaction * sizeof(unsigned short int *));

    while ((read = getline(&line, &len, fp)) != -1){
    	// printf("%s", line);
		transactions[lines] = (unsigned short int *) malloc(max_items_in_transaction * sizeof(unsigned short int));
    	
    	count = 0;

    	ln = strtok(line, " ");
    	if (ln != NULL){
    			//unsigned short int a = (unsigned short int) strtoul(ln, NULL, 0);
    			transactions[lines][count++] = (unsigned short int) strtoul(ln, NULL, 0);
    	}
    	
    	while (ln != NULL){
    		// printf("%s ", ln);
    		ln = strtok(NULL, " ");
    		if (ln != NULL){
    			transactions[lines][count++] = (unsigned short int) strtoul(ln, NULL, 0);
    		}

    	}
    	transactions[lines][count] = NULL;
   		lines++;
    }
    fclose(fp);
    transactions[lines] = NULL;

    for(int i = 0; i < lines; i++){
    	int j = 0;
    	unsigned short int *temp = transactions[i];
   		while(temp != NULL && temp[j] != NULL){
   			printf("%x, ", temp[j]);
   			j++;
   		}
   		//printf("\n");
    }
    printf("%d\n", lines);

    /////////////////////////////////////////////////////////////////////////////////////

    

    return 0;

}
