
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE


#define max_items_in_transaction 128
#define max_num_of_transaction 1000000
#define max_unique_items 12000
#define BLOCK_SIZE 1024

__global__ void makeFlist(unsigned int *d_trans_offset, unsigned int *d_transactions, unsigned int *d_flist, unsigned int num_transactions, unsigned int num_items_in_transactions){

	__shared__ unsigned int private_items[max_unique_items];

	int tx = threadIdx.x;
	int index = tx + blockDim.x * blockIdx.x;
	int location_x;

	for (int i = 0; i < ceil(max_unique_items / (1.0 * BLOCK_SIZE)); i++){
		location_x = tx + i * blockDim.x;
		if ( location_x < max_unique_items)
			private_items[location_x] = 0;
	}

	__syncthreads();
	
	//int item_ends = 0;
	
	// if (tx == (num_transactions - 1)){
	// 	item_ends = num_items_in_transactions;
	// }else{
	// 	item_ends = d_trans_offset[index + 1];
	// }
	// //int j = 0;
	// for(int i = d_trans_offset[index]; i < item_ends; i++){
	// 	if (d_transactions[i] < max_unique_items)
	// 		atomicAdd(&private_items[d_transactions[i]], 1);
	// 	//j = d_transactions[i];
	// }
	if (index < num_items_in_transactions && d_transactions[index] < max_unique_items)
		atomicAdd(&private_items[d_transactions[index]], 1);

	__syncthreads();

	for (int i = 0; i < ceil(max_unique_items / (1.0 * BLOCK_SIZE)); i++){
		location_x = tx + i * blockDim.x;
		if ( location_x < max_unique_items)
			atomicAdd(&d_flist[location_x], private_items[location_x]);
	}
	__syncthreads();

}
