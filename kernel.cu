
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

#define max_items_in_transaction 128
#define max_num_of_transaction 1000000

#define max_unique_items 8192

#define BLOCK_SIZE 512

__global__ void makeFlist(unsigned int *d_trans_offset, unsigned int *d_transactions , unsigned int num_transactions, unsigned int num_items_in_transactions){

	__shared__ unsigned int private_items[max_unique_items];

	int tx = threadIdx.x + blockDim.x * blockIdx.x;
	int location_x;

	for (int i = 0; i < ceil(max_unique_items / (1.0 * blockDim.x)); i++){
		location_x = tx + i * blockDim.x;
		if ( location_x < max_unique_items)
			private_items[location_x] = 0;
	}
	__syncthreads();

	
	int item_ends = 0;
	
	if (tx == (num_transactions - 1)){
		item_ends = num_items_in_transactions;
	}else{
		item_ends = d_trans_offset[tx + 1];
	}
	//int j = 0;
	for(int i = d_trans_offset[tx]; i < item_ends; i++){
		if (d_transactions[i] >=0 && d_transactions[i] < max_unique_items)
			atomicAdd(&private_items[0], 1);
		//j = d_transactions[i];
	}
}
