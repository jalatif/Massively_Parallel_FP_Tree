
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

#define max_items_in_transaction 128
#define max_num_of_transaction 1000000

#define max_unique_items 16384

#define BLOCK_SIZE 1024

__global__ void makeFlist(unsigned int *d_trans_offset, unsigned short int *d_transactions , unsigned int num_transactions, unsigned int num_items_in_transactions){

	__shared__ unsigned short int private_items[max_unique_items];

	int tx = threadIdx.x + blockDim.x * blockIdx.x;
	
	int item_ends = 0;
	
	if (tx == (num_transactions - 1)){
		item_ends = num_items_in_transactions;
	}else{
		item_ends = d_trans_offset[tx + 1];
	}

	for(int i = d_trans_offset[tx]; i < item_ends; i++){
		atomicAdd((int*)&private_items[d_transactions[i]], 1);
	}
}
