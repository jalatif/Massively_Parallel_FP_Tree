#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <stdlib.h>

__device__ inline unsigned int uatomicAdd(unsigned int* address, int incr){

	int expected = *address;
	int old_val = atomicCAS((int*) address, expected, expected + incr);

	while (old_val != expected){
		expected = old_val;
		old_val = atomicCAS((int*) address, expected, expected + incr);
	}
	return old_val;
}

__global__ void histogram_kernel(unsigned int *input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins){

	__shared__ short int private_items[];

	int tx = threadIdx.x;
	int current = tx + blockDim.x * blockIdx.x;
	int location_x;

	for (int i = 0; i < ceil(num_bins / (1.0 * blockDim.x)); i++){
		location_x = tx + i * blockDim.x;
		if ( location_x < num_bins)
			private_histogram[location_x] = 0;
	}
	__syncthreads();

	if (current < num_elements && input[current] < num_bins){
		atomicAdd((unsigned int*)(&private_histogram[input[current]]), 1);
	}

	__syncthreads();

	for (int i = 0; i < ceil(num_bins / (1.0 * blockDim.x)); i++){
		location_x = tx + i * blockDim.x;
		if (location_x < num_bins)
			atomicAdd((int*)(&bins[location_x]), private_histogram[location_x]);
	}
}





/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements,
        unsigned int num_bins) {

    // INSERT CODE HERE
	unsigned int block_size = 512;
    dim3 grid_dim, block_dim;

    block_dim.x = block_size; 
    block_dim.y = 1; block_dim.z = 1;
	
	grid_dim.x = ceil(num_elements / (1.0 * block_size)); 
	grid_dim.y = 1; grid_dim.z = 1;

	size_t private_hist_size = num_bins * sizeof(unsigned int);
	histogram_kernel<<<grid_dim, block_dim, private_hist_size>>>(input, bins, num_elements, num_bins);
}


/// Dynamic Shared memory
///
